#include "hip/hip_runtime.h"
#include "ctdetLayer.h"
#include "utils.h"
 
// BLOCK = 512
dim3 cudaGridSize(uint n)
{
    uint k = (n - 1) /BLOCK + 1;
    uint x = k ;
    uint y = 1 ;
    if (x > 65535 )
    {
        x = ceil(sqrt(x));
        y = (n - 1 )/(x*BLOCK) + 1;
    }
    dim3 d = {x,y,1} ;
    return d;
}
__device__ float Logist(float data){ return 1./(1. + exp(-data)); }

__global__ void CTdetforward_kernel(const float *hm, const float *reg,const float *wh ,
        float *output,const int w,const int h,const int classes,const int kernel_size,const float visthresh  ) {
    int idx = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (idx >= w * h * classes) return;
    int padding = (kernel_size - 1) / 2;
    int offset = -padding;
    int stride = w * h;
    int grid_x = idx % w;
    int grid_y = (idx / w) % h;
    int cls = idx/w/h ;
    int  l, m;
    int reg_index = idx - cls*stride;
    float c_x, c_y;
    float objProb = Logist(hm[idx]);
    if (objProb > visthresh) {
        float max = -1;
        int max_index = 0;
        for (l = 0; l < kernel_size; ++l)
            for (m = 0; m < kernel_size; ++m) {
                int cur_x = offset + l + grid_x;
                int cur_y = offset + m + grid_y;
                int cur_index = cur_y * w + cur_x + stride * cls;
                int valid = (cur_x >= 0 && cur_x < w && cur_y >= 0 && cur_y < h);
                float val = (valid != 0) ? Logist(hm[cur_index]) : -1;
                max_index = (val > max) ? cur_index : max_index;
                max = (val > max) ? val : max;
            }

        if(idx == max_index){
            int resCount = (int) atomicAdd(output, 1);
            //printf("%d",resCount);
            char *data = (char *) output + sizeof(float) + resCount * sizeof(Detection);
            Detection *det = (Detection *) (data);
            c_x = grid_x + reg[reg_index];
            c_y = grid_y + reg[reg_index + stride];
            det->bbox.x1 = (c_x - wh[reg_index] / 2) * 4;
            det->bbox.y1 = (c_y - wh[reg_index + stride] / 2) * 4;
            det->bbox.x2 = (c_x + wh[reg_index] / 2) * 4;
            det->bbox.y2 = (c_y + wh[reg_index + stride] / 2) * 4;
            det->classId = cls;
            det->prob = objProb;
        }
    }
}

/*
__global__ void CTdddforward_kernel(const float *hm, const float *reg,const float *wh ,
        float *output,const int w,const int h,const int classes,const int kernel_size,const float visthresh  ) {
    int idx = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (idx >= w * h * classes) return;
    int padding = (kernel_size - 1) / 2;
    int offset = -padding;
    int stride = w * h;
    int grid_x = idx % w;
    int grid_y = (idx / w) % h;
    int cls = idx/w/h ;
    int  l, m;
    int reg_index = idx - cls*stride;
    float c_x, c_y;
    float objProb = Logist(hm[idx]);
    if (objProb > visthresh) 
    {
        float max = -1;
        int max_index = 0;
        for (l = 0; l < kernel_size; ++l)
            for (m = 0; m < kernel_size; ++m) {
                int cur_x = offset + l + grid_x;
                int cur_y = offset + m + grid_y;
                int cur_index = cur_y * w + cur_x + stride * cls;
                int valid = (cur_x >= 0 && cur_x < w && cur_y >= 0 && cur_y < h);
                float val = (valid != 0) ? Logist(hm[cur_index]) : -1;
                max_index = (val > max) ? cur_index : max_index;
                max = (val > max) ? val : max;
            }

        if(idx == max_index){
            int resCount = (int) atomicAdd(output, 1);
            printf("%d",resCount);
            
            char *data = (char *) output + sizeof(float) + resCount * sizeof(Detection);
            Detection *det = (Detection *) (data);
            c_x = grid_x + reg[reg_index];
            c_y = grid_y + reg[reg_index + stride];
            det->bbox.x1 = (c_x - wh[reg_index] / 2) * 4;
            det->bbox.y1 = (c_y - wh[reg_index + stride] / 2) * 4;
            det->bbox.x2 = (c_x + wh[reg_index] / 2) * 4;
            det->bbox.y2 = (c_y + wh[reg_index + stride] / 2) * 4;
            det->classId = cls;
            det->prob = objProb;
        }
    }
}
*/

__global__ void CTdddforward_kernel(const float *hm, const float *reg,const float *wh , const float *dim , const float *rot , const float *dep ,
        float *output,const int w,const int h,const int classes,const int kernel_size,const float visthresh  ) {
    int idx = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (idx >= w * h * classes) return;
    int padding = (kernel_size - 1) / 2;
    int offset = -padding;
    int stride = w * h;
    int grid_x = idx % w;
    int grid_y = (idx / w) % h;
    int cls = idx/w/h ;
    int  l, m;
    int reg_index = idx - cls*stride;
    float c_x, c_y;
    float objProb = Logist(hm[idx]);
    if (objProb > visthresh) 
    {
        float max = -1;
        int max_index = 0;
        for (l = 0; l < kernel_size; ++l)
            for (m = 0; m < kernel_size; ++m) {
                int cur_x = offset + l + grid_x;
                int cur_y = offset + m + grid_y;
                int cur_index = cur_y * w + cur_x + stride * cls;
                int valid = (cur_x >= 0 && cur_x < w && cur_y >= 0 && cur_y < h);
                float val = (valid != 0) ? Logist(hm[cur_index]) : -1;
                max_index = (val > max) ? cur_index : max_index;
                max = (val > max) ? val : max;
            }

        if(idx == max_index){
            int resCount = (int) atomicAdd(output, 1);
            printf("%d",resCount);
            
            char *data = (char *) output + sizeof(float) + resCount * sizeof(Detection);
            Detection *det = (Detection *) (data);
            c_x = grid_x + reg[reg_index];
            c_y = grid_y + reg[reg_index + stride];
            det->bbox.x1 = (c_x - wh[reg_index] / 2) * 4;
            det->bbox.y1 = (c_y - wh[reg_index + stride] / 2) * 4;
            det->bbox.x2 = (c_x + wh[reg_index] / 2) * 4;
            det->bbox.y2 = (c_y + wh[reg_index + stride] / 2) * 4;
            det->classId = cls;
            det->prob = objProb;
            det->size3d.h = dim[reg_index];
            det->size3d.w = dim[reg_index + stride];
            det->size3d.l = dim[reg_index + stride + stride];
            det->rot.ang1 = rot[reg_index];
            det->rot.ang2 = rot[reg_index + stride];
            det->rot.ang3 = rot[reg_index + stride + stride];
            det->rot.ang4 = rot[reg_index + stride + stride + stride];
            det->rot.ang5 = rot[reg_index + stride + stride + stride + stride];
            det->rot.ang6 = rot[reg_index + stride + stride + stride + stride + stride];
            det->rot.ang7 = rot[reg_index + stride + stride + stride + stride + stride + stride];
            det->rot.ang8 = rot[reg_index + stride + stride + stride + stride + stride + stride + stride];   
            det->dep = 1. / (Logist(dep[reg_index])+1e-6) -1.;
            // det->dep = dep[reg_index];
        }
    }
}



__global__ void CTfaceforward_kernel(const float *hm, const float *wh,const float *reg,const float* landmarks,
                                    float *output,const int w,const int h,const int classes,const int kernel_size,const float visthresh  ) {
    int idx = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (idx >= w*h*classes) return;
    int padding = (kernel_size-1)/2;
    int offset = - padding;
    int stride = w * h;
    int grid_x = idx % w;
    int grid_y = (idx / w) % h;
    int cls = idx/w/h ;
    int reg_index = idx - cls*stride;
    int l,m,mark_id;
    float c_x,c_y,scale_w,scale_h;

    float objProb = hm[idx];
    float max=-1;
    int max_index =0;
    if(objProb > visthresh){
        for(l=0 ;l < kernel_size ; ++l)
            for(m=0 ; m < kernel_size ; ++m){
                int cur_x = offset + l + grid_x;
                int cur_y = offset + m + grid_y;
                int cur_index = cur_y * w + cur_x + stride*cls;
                int valid = (cur_x>=0 && cur_x < w && cur_y >=0 && cur_y <h );
                float val = (valid !=0 ) ? hm[cur_index]: -1;
                max_index = (val > max) ? cur_index : max_index;
                max = (val > max ) ?  val: max ;
            }
        if(idx == max_index){
            int resCount = (int)atomicAdd(output,1);
            //printf("%d",resCount);
            char* data = (char * )output + sizeof(float) + resCount*sizeof(Detection);
            Detection* det =  (Detection*)(data);
            c_x = (grid_x + reg[reg_index+stride] + 0.5)*4 ; c_y  = (grid_y + reg[reg_index] + 0.5) * 4;
            scale_w =  expf(wh[reg_index+stride]) * 4 ; scale_h  = expf(wh[reg_index]) * 4;
            det->bbox.x1 = c_x - scale_w/2;
            det->bbox.y1 = c_y - scale_h/2 ;
            det->bbox.x2 = c_x + scale_w/2;
            det->bbox.y2 = c_y + scale_h/2;
            det->prob = objProb;
            det->classId = cls;
            for(mark_id=0 ; mark_id < 5 ; ++ mark_id){
                det->marks[mark_id].x = det->bbox.x1 + landmarks[reg_index + (2*mark_id+1)*stride]*scale_w;
                det->marks[mark_id].y = det->bbox.y1 + landmarks[reg_index + (2*mark_id)*stride]*scale_h;
            }
        }
    }
}


void CTdetforward_gpu(const float *hm, const float *reg,const float *wh ,float *output,
                      const int w,const int h,const int classes,const int kernerl_size, const float visthresh ){
    uint num = w * h * classes;
    CTdetforward_kernel<<<cudaGridSize(num),BLOCK>>>(hm,reg,wh,output,w,h,classes,kernerl_size,visthresh);
}

void CTfaceforward_gpu(const float *hm, const float *wh,const float *reg,const float* landmarks,float *output,
                      const int w,const int h,const int classes,const int kernerl_size, const float visthresh ){
    uint num = w * h * classes;
    CTfaceforward_kernel<<<cudaGridSize(num),BLOCK>>>(hm,wh,reg,landmarks,output,w,h,classes,kernerl_size,visthresh);
}

void CTdddforward_gpu(const float *hm, const float *dep, const float *rot, const float *dim, const float *wh, const float *reg, float *output,
                       const int w,const int h,const int classes,const int kernerl_size, const float visthresh ){
    uint num = w * h * classes;
    // w and h are the sizes of feature map
    // num = 96*320*3=92160
    // BLOCK = 512
    // std::cout<<"num"<<num<<std::endl;
    //CTdddforward_kernel<<<cudaGridSize(num),BLOCK>>>(hm,reg,wh,output,w,h,classes,kernerl_size,visthresh);
    CTdddforward_kernel<<<cudaGridSize(num),BLOCK>>>(hm,reg,wh,dim,rot,dep,output,w,h,classes,kernerl_size,visthresh);
}
